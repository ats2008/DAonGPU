#include "hip/hip_runtime.h"
#include "gpuDAVertexer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include "stdio.h"

//#define FULL_DEVICE_DEBUG

namespace gpuDAVertexer {

__global__ void demoKernel(ZTrackSoA * tracks,int n)
{
    if(!tracks) printf("null as trks");
    int idx =threadIdx.x + blockIdx.x*blockDim.x;
    //printf("HI HI !! in device %d  idx = %d %f \n",n,idx,tracks[15].pt[0]);
    if (idx<n)
    {
        printf("On Devise !! [%d + %d * %d]  : track[%d].pt[0] = %f \n",threadIdx.x,blockIdx.x,blockDim.x,idx,tracks[idx].pt[0]);
    }
    else
    {
        printf("On Devise !! [%d + %d * %d] = %d\n ",threadIdx.x,blockIdx.x,blockDim.x,idx);
    }
}

__global__ void initializeWorspace(Workspace * wrkspace)
{

    auto idx= blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < MAXTRACKS*MAXVTX)
    {
        wrkspace->pik[idx]=1.0;
        wrkspace->pik_numer[idx]=0.0;
        wrkspace->pik_denom[idx]=0.0;

        wrkspace->zk_delta[idx]=1e8;
        wrkspace->zk_numer[idx]=0.0;
        wrkspace->zk_denom[idx]=0.0;
    }
    if(idx <MAXVTX)
    {
        wrkspace->zVtx[idx]=1e9;
        wrkspace->rhok[idx]=1;
        wrkspace->dauterMap[idx]=-1;
    }
    if(idx==0)
    {
        wrkspace->nVertex=0;
        wrkspace->temp_nVertex=0;
        wrkspace->betaFactor=1.5;
        wrkspace->betaSplitMax=0.24;
    	wrkspace->rho_denom=1.0;
        wrkspace->maxDZforMerge=0.01;
    }

}

__global__ void loadTracks(ZTrackSoA * tracks,Workspace * wrkspace)
{

    auto idx= blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < tracks->ntrks)
    {
        wrkspace->zt[idx]=tracks->zt[idx];
        wrkspace->dz2[idx]=tracks->dz2[idx];
        wrkspace->pi[idx]=1.0;
    }
    if(idx==0)
    {
        if(tracks->ntrks % 2 and false)
        {
            wrkspace->zt[idx]=0.0;
            wrkspace->dz2[idx]=1e9;
            wrkspace->pi[idx]=0.0;
            wrkspace->nTracks = tracks->ntrks+1;
        }
        else
        {
            wrkspace->nTracks=tracks->ntrks;
        }
    }

}

/// ==================================================//
__device__ void sumBlock_with_shfl_down_gid_DF(float *in, float *out, int Ntracks,int Nvertices)
{

    if(threadIdx.x<Nvertices)
    {
        int vtxId=threadIdx.x*Ntracks;
        for (int offset =1 ; offset <Ntracks; offset ++) // (blockSize/2)+1
        {
            out[vtxId] += in[vtxId+offset] ;
        }
#ifdef FULL_DEVICE_DEBUG
        printf("sumBlock_with_shfl_down_gid_DF threadIdx.x = %d, out = %f \n",threadIdx.x,out[vtxId]);
#endif
    }


}



__device__ void sumBlock_with_loop_DF(float *in, float *out,int numVertices, int Ntracks)
{
    int tid = threadIdx.x;
    if(tid<Ntracks)
    {
        int off ;
        //out[tid]=0.0;
        for (int offset = 1 ; offset<numVertices; offset++ )
        {
            off = Ntracks*offset ;
            out[tid] += in[tid+off];
        }
#ifdef FULL_DEVICE_DEBUG
    printf("sumBlock_with_loop_DF out[%d] = %f \n",tid,out[tid]);
#endif
    }
}

__global__ void kernel_findFreeEnergyPartA(float *FEnergyA,float * zi, float *zVtx,float* sig,float beta,int CurrentVtx,int N )
{
    int idx = threadIdx.x;
    int bid = blockIdx.x; // block Id
    int gid = blockIdx.x * blockDim.x + idx;
    int TotalSize = N*CurrentVtx; //  nTracks * nVertex

    if (gid < TotalSize) {
        FEnergyA[gid] =  expf( -beta*((zi[idx]-zVtx[bid])*(zi[idx]-zVtx[bid])/sig[idx] ));
//    printf("gid = %d , dz = %f - %f = %f , FEnergyA[gid] = %f\n",gid,zi[idx],zVtx[bid],zi[idx]-zVtx[bid], FEnergyA[gid]);
    }
}
__global__ void kernel_findFreeEnergyPartB(float * FEnergyA, float beta, int currVtxCount,int N)
{

    auto fEnergy=0.0;
    for(int i=0; i<N; i++)
    {
        auto asum=0.0;
        for(int j=0; j<currVtxCount; j++)
        {

            asum+=FEnergyA[i+j*N];
        }
//	printf("( %d , %f ,%f )",i,asum,beta);
        fEnergy-=logf(asum >1e-20 ? asum : 1.0 )/beta;

    }
    printf("\n$(beta , Free Energy) : ,%f,%f\n",beta,fEnergy);
}



__device__ void kernel_p_ik_numDenom_DF( float *p_ik, float *p_ik_denom,float * rhok, float *z_i, float *z_k0,   float *sig, float beta, int Ntracks, int numberOfvertex)
{

    auto strideLen = blockDim.x;

    for(auto tid=threadIdx.x; tid<Ntracks; tid+=strideLen)
    {
        for(auto vid=0; vid<numberOfvertex; vid++)
        {
            auto gid = vid*Ntracks + tid ;
            p_ik[gid] =  expf( -beta*(((z_i[tid]-z_k0[vid])*(z_i[tid]-z_k0[vid]))/(sig[tid]*sig[tid]*sig[tid]*sig[tid])) );
            p_ik_denom[gid] = p_ik[gid]*rhok[vid];
#ifdef FULL_DEVICE_DEBUG
            printf("pikNum DEVICE : gid = %d , dz = %f - %f = %f,rhok[%d] = %f , pik = %f, pik_den =%f \n",gid,z_i[tid],z_k0[vid],z_i[tid]-z_k0[vid],\\
	    					vid,rhok[vid], p_ik[gid],p_ik_denom[gid]);
#endif
            //auto x =  expf( -beta*(((z_i[tid]-z_k0[vid])*(z_i[tid]-z_k0[vid]))/(sig[tid]*sig[tid]*sig[tid]*sig[tid])) );
            //printf("DEVICE : gid = %d , dz = %f - %f = %f , pik = %f\n",gid,z_i[tid],z_k0[vid],z_i[tid]-z_k0[vid], x);
        }
    }
}


__device__ void kernel_p_ik_DF( float *p_ik, float *p_ik_den, int Ntracks, int numberOfvertex )
{

    auto strideLength=blockDim.x;

    for(auto tid=threadIdx.x; tid<Ntracks; tid+=strideLength)
    {
        if(tid>Ntracks) break;

        for(auto vid=0; vid<numberOfvertex; vid++)
        {
            auto gid=tid+vid*Ntracks;
#ifdef FULL_DEVICE_DEBUG
            auto oldval = p_ik[gid];
#endif
            if (p_ik_den[tid] > 1.e-45)
            {
                p_ik[gid] =  p_ik[gid]/p_ik_den[tid] ;
            }
            else
            {
                p_ik[gid] =  0.000 ;
            }
#ifdef FULL_DEVICE_DEBUG
            printf("pik DIVICE : pik[%d] = pik_[%d] / p_ik_den[%d] = %f/ %f = %f\n",\\
                   gid,gid,tid,oldval,p_ik_den[tid],p_ik[gid]);
#endif

        }

    }
}


__device__ void kernel_z_ik_num_DF( float *p_ik, float *z_ik_num, float *p_i, float *z_i, float *sig, int Ntracks, int numberOfvertex )
{

    auto strideLength=blockDim.x;
    for(auto tid=threadIdx.x; tid<Ntracks; tid+=strideLength)
    {
        if(tid>Ntracks) break;

        for(auto vid=0; vid<numberOfvertex; vid++)
        {
            auto gid=tid+vid*Ntracks;
            z_ik_num[gid] = p_i[tid]*p_ik[gid]*z_i[tid]/(sig[tid]*sig[tid]);
#ifdef FULL_DEVICE_DEBUG
            printf("zikNum DEVICE : z_ik_num[%d] = %f ,tid  = %d , z_i[tid]*p_i[tid]*p_ik[gid]/sig[tid]  = %f*%f*%f/%f^2 ,	sig[tid] = %f \n",gid,z_ik_num[gid],tid,z_i[tid],p_i[tid],p_ik[gid],sig[tid],sig[tid]);
#endif
        }
    }

}




__device__ void kernel_z_ik_den_DF( float *p_ik, float *z_ik_den, float *p_i, float *z_i, float *sig, int Ntracks, int numberOfvertex )
{

    auto strideLength=blockDim.x;
    for(auto tid=threadIdx.x; tid<Ntracks; tid+=strideLength)
    {
        if(tid>Ntracks) break;

        for(auto vid=0; vid<numberOfvertex; vid++)
        {
            auto gid=tid+vid*Ntracks;
            z_ik_den[gid] = p_i[tid]*p_ik[gid]/(sig[tid]*sig[tid]);
#ifdef FULL_DEVICE_DEBUG
            printf("zikDen DEVICE : z_ik_den[%d] = %f ,tid  = %d , p_i[tid]*p_ik[gid]/sig[tid]^2 = %f*%f/%f \n",gid,z_ik_den[gid],tid,p_i[tid],p_ik[gid],sig[tid]);
#endif
            //	auto x = p_i[tid]*p_ik[gid]/(sig[tid]*sig[tid]);
            //     printf("DEVICEL : z_ik_den[%d] = %f ,idx  = %d , p_i[idx]*p_ik[gid]/sig[idx]^2 = %f*%f/%f \n",gid,x,idx,p_i[idx],p_ik[gid],sig[idx]);

        }

    }
}

__device__ void  kernel_z_ik_DF(float * zk_numer,float * zk_denom,float * zDelta,float* zVtx,int ntraks,int currVtxCount )
{
    if(threadIdx.x < currVtxCount)
    {
        auto newZ=zk_numer[threadIdx.x*ntraks]/(1e-20 + zk_denom[threadIdx.x*ntraks]);
        zDelta[threadIdx.x] = abs(zVtx[threadIdx.x] - newZ);
        zVtx[threadIdx.x]   = newZ;

#ifdef FULL_DEVICE_DEBUG
        printf("zik DEVICE : setting Z[%d] = %f ,delta[%d] = %f ,numer = %f , deno = %f \n",\\
               threadIdx.x,zVtx[threadIdx.x],threadIdx.x,zDelta[threadIdx.x], zk_numer[threadIdx.x*ntraks],zk_denom[threadIdx.x]);
#endif
    }

}

__device__ void kernel_tc_num_DF( float *T_num, float *z_i, float *zVtx, float *p,float *p_ik, float *sig, int Ntracks,int numberOfvertex)
{
    auto strideLength=blockDim.x;
    for(auto tid=threadIdx.x; tid<Ntracks; tid+=strideLength)
    {
        if(tid>Ntracks) break;
        for(auto vid=0; vid<numberOfvertex; vid++)
        {
            auto gid=tid+vid*Ntracks;
            T_num[gid] = p[tid]*p_ik[gid]*((z_i[tid]-zVtx[vid])*(z_i[tid]-zVtx[vid]))/(sig[tid]*sig[tid]);
#ifdef FULL_DEVICE_DEBUG
            printf("tcNum DEVICE : tid = %d, p[tid] =%f , z_i[tid] = %f ,zVtx[%d] =%f ,sig[tid] =%f , Tnum[%d] = %f \n",tid,p[tid],z_i[tid],vid,zVtx[vid],sig[tid],gid,T_num[gid]);
#endif
        }
    }
}

__device__ void  kernel_tc_k_DF(float * tc_numer,float * tc_denom,float* tc,int ntraks,int currVtxCount )
{
    if(threadIdx.x < currVtxCount)
    {
        tc[threadIdx.x] = 2.0*tc_numer[threadIdx.x*ntraks]/(1e-20 + tc_denom[threadIdx.x*ntraks]);
#ifdef FULL_DEVICE_DEBUG
        printf("tcK DEVICE setting tc[%d] = %f , numer = %f , deno = %f \n",\\
               threadIdx.x,tc[threadIdx.x], tc_numer[threadIdx.x*ntraks],tc_denom[threadIdx.x]);
#endif
    }
}
__device__ void check_ifThermalized(float * deltas,float deltaTol,int *hasThermalized,int currVtxCount)
{
    if(threadIdx.x<currVtxCount)
    {
        if(deltas[threadIdx.x]>deltaTol)
        {
            atomicOr(hasThermalized,1);
        }
    }

}

// probably pass on the z2 avg and spit approximating the xluster to be 2 gaussians
__device__ void kernel_z_k_spliting_DF(float temp,float *z_k,float *rhok,int * dauterMap, float * tc_clusters,uint32_t *cur_NV)
{
    /*
       This kernel take the vertex list and split the last vertex into z-delta,z+delta (delta between 0 and 1.0)
    */

    auto tid= threadIdx.x;
    if (tid >= *cur_NV)
        return;
    if(temp>tc_clusters[tid])
    {
        printf("Checking for vertex %d at T= %f  and Tc = %f \n ",tid,temp,tc_clusters[tid]);
        return;
    }
    
    auto idx =  atomicAdd(cur_NV,1);

    float z_k_aux =z_k [tid];

    // calculate the deltaZk
    /*

    auto deltaZk = sqrt( <Z^2>_k - (z_k)^2 )
    */

    //auto deltaZk  = abs(0.2*z_k[tid]);
    auto deltaZk  =0.01;
    z_k[tid] = z_k_aux - deltaZk;
    z_k[idx] = z_k_aux + deltaZk;
    dauterMap[tid]=idx;
    
    rhok[tid]*=0.5;
    rhok[idx] =rhok[tid];
    printf("Splitting vertex %d at T= %f  and Tc = %f, delta = %f z_old = %f z_new[%d] = %f\n ",tid,temp,tc_clusters[tid],deltaZk,z_k[tid],idx,z_k[idx]);
}
__global__ void kernel_z_k_spliting_DF_DK(float temp,float *z_k,float *rhok,int *dauterMap, float * tc_clusters,uint32_t *cur_NV)
{

    kernel_z_k_spliting_DF(temp,z_k, rhok,dauterMap,tc_clusters,cur_NV) ;
}

__device__ void kernel_rho_k_num_DF( float *p_ik, float *rho_k_num, float *p_i, int Ntracks, int numberOfvertex )
{

    auto strideLength=blockDim.x;
    for(auto tid=threadIdx.x; tid<Ntracks; tid+=strideLength)
    {
        if(tid>Ntracks) break;

        for(auto vid=0; vid<numberOfvertex; vid++)
        {
            auto gid=tid+vid*Ntracks;
            rho_k_num[gid] = p_i[tid]*p_ik[gid];
#ifdef FULL_DEVICE_DEBUG
            printf("rhok_num DEVICE : rho_k_num[%d] = %f ,tid  = %d , p_i[tid]*p_ik[gid]  = %f*%f  \n",gid,rho_k_num[gid],tid,p_i[tid],p_ik[gid]);
#endif
        }
    }

}

__device__ void  kernel_rho_k_DF(float * rhok_numer,float rhok_denom,float* rhok,int ntraks,int currVtxCount )
{
    if(threadIdx.x < currVtxCount)
    {
        rhok[threadIdx.x]   = rhok[threadIdx.x]*rhok_numer[threadIdx.x*ntraks]/(1e-20 + rhok_denom);

#ifdef FULL_DEVICE_DEBUG
        printf("rhok DEVICE : setting rhok[%d] = %f  , numer = %f , deno = %f \n",\\
               threadIdx.x,rhok[threadIdx.x],rhok_numer[threadIdx.x*ntraks],rhok_denom);
#endif
    }

}


/// =================================================//

// device functions might also be made inline , have to check if we will get any performance inprovements in this
// The calculation of Z, the Eik calculation has  space complexity of numTracks*numVertices
// In the original DA code they have arranged tracks in the acending Z and only the tracks which are close to a vertex goes into the ccalculation
// the farther tracks will only contibute very less since its supressed by exp (-Eik ) , we may have to also incorporate it after a basic working code is ready.


__global__ void initializeDAvertexReco( Workspace *wrkspace  )
{

    auto N=wrkspace->nTracks;
    if(threadIdx.x==0)
        printf("at initialization N  = %d \n",N);
    auto CurrentNvetex = 1;
    //      >>>>>>>>>KERNELs for ZVtx Update<<<<<<<<<
    kernel_z_ik_num_DF(wrkspace->pik, wrkspace->zk_numer, wrkspace->pi,wrkspace->zt,wrkspace->dz2, N, CurrentNvetex);
    hipDeviceSynchronize();
    kernel_z_ik_den_DF(wrkspace->pik, wrkspace->zk_denom, wrkspace->pi, wrkspace->zt, wrkspace->dz2, N, CurrentNvetex);
    __syncthreads();
    hipDeviceSynchronize();

    sumBlock_with_shfl_down_gid_DF(wrkspace->zk_numer, wrkspace->zk_numer, N,CurrentNvetex);
    hipDeviceSynchronize();
    sumBlock_with_shfl_down_gid_DF(wrkspace->zk_denom, wrkspace->zk_denom, N,CurrentNvetex);
    __syncthreads();
    hipDeviceSynchronize();

    kernel_z_ik_DF(wrkspace->zk_numer, wrkspace->zk_denom,wrkspace->zk_delta,wrkspace->zVtx, N, CurrentNvetex);
    hipDeviceSynchronize();
    wrkspace->nVertex=1;

    //      >>>>>>>>>KERNEL for T finding <<<<<<<<<
    kernel_tc_num_DF(wrkspace->tc_numer,wrkspace->zt,wrkspace->zVtx,wrkspace->pi,wrkspace->pik,wrkspace->dz2,\\
                     N,CurrentNvetex);
    hipDeviceSynchronize();
    __syncthreads();

    sumBlock_with_shfl_down_gid_DF(wrkspace->tc_numer,wrkspace->tc_numer,N, CurrentNvetex);
    hipDeviceSynchronize();
    __syncthreads();

    kernel_tc_k_DF(wrkspace->tc_numer,wrkspace->zk_denom,wrkspace->tc,N,CurrentNvetex);
    hipDeviceSynchronize();
    __syncthreads();

    if(threadIdx.x==0)
    {
        wrkspace->beta=1.0/(1e-9 + (wrkspace->tc)[0] );
        wrkspace->rhok[0]=1.0;
        wrkspace->rho_denom=N;
        printf(" workspace rhok[0] = %f\f, beta set to %f ( 1.0/%f  ) bets split max : %f \n",wrkspace->rhok[0],wrkspace->beta,wrkspace->tc[0],wrkspace->betaSplitMax);
    }

}

__device__ void updateTrackToVertexProbablilities(Workspace * wrkspace)
{
    if(threadIdx.x==0)
        printf("In the updateTrackToVertexProbablilities\n");

//      >>>>>>>>> KERNELs for  kernel_p_ik <<<<<<<<<
    auto N=wrkspace->nTracks;
    auto CurrentNvetex=wrkspace->nVertex;
    if(threadIdx.x==0)
        printf("with N = %d , CurrentNvetex = %d \n",N,CurrentNvetex);

    kernel_p_ik_numDenom_DF(wrkspace->pik,wrkspace->pik_denom,wrkspace->rhok,wrkspace->zt,wrkspace->zVtx, wrkspace->dz2, wrkspace->beta, N, CurrentNvetex);
    __syncthreads();

    sumBlock_with_loop_DF(wrkspace->pik_denom,wrkspace->pik_denom,CurrentNvetex,N);
    __syncthreads();

    kernel_p_ik_DF(wrkspace->pik,wrkspace->pik_denom,N,CurrentNvetex);
    __syncthreads();
}

__device__ void updateVertexPositions(Workspace *wrkspace)
{
    auto N=wrkspace->nTracks;
    auto CurrentNvetex=wrkspace->nVertex;

    if(threadIdx.x==0)
        printf("In the updateVertexPositions wit %d vertexes \n",wrkspace->nVertex);
    //      >>>>>>>>>KERNELs for ZVtx Update<<<<<<<<<

    kernel_z_ik_num_DF(wrkspace->pik, wrkspace->zk_numer, wrkspace->pi, wrkspace->zt,wrkspace->dz2, N, CurrentNvetex);
    kernel_z_ik_den_DF(wrkspace->pik, wrkspace->zk_denom, wrkspace->pi, wrkspace->zt, wrkspace->dz2, N, CurrentNvetex);
    __syncthreads();

    sumBlock_with_shfl_down_gid_DF(wrkspace->zk_numer, wrkspace->zk_numer, N,CurrentNvetex);
    sumBlock_with_shfl_down_gid_DF(wrkspace->zk_denom, wrkspace->zk_denom, N,CurrentNvetex);
    __syncthreads();

    kernel_z_ik_DF(wrkspace->zk_numer, wrkspace->zk_denom,wrkspace->zk_delta,wrkspace->zVtx, N, CurrentNvetex);
}




__device__ void updateVertexWeights(Workspace * wrkspace)
{
    if(threadIdx.x==0)
        printf("In the updateVertexWeights\n");

    auto N=wrkspace->nTracks;
    auto CurrentNvetex=wrkspace->nVertex;

    kernel_rho_k_num_DF(wrkspace->pik, wrkspace->rhok_numer, wrkspace->pi,N, CurrentNvetex);
    __syncthreads();
    
    sumBlock_with_shfl_down_gid_DF(wrkspace->rhok_numer, wrkspace->rhok_numer, N,CurrentNvetex);
    __syncthreads();

    kernel_rho_k_DF(wrkspace->rhok_numer, wrkspace->rho_denom,wrkspace->rhok, N, CurrentNvetex);

}

__device__  void updateClusterCriticalTemperatures(Workspace *wrkspace)
{
    auto N=wrkspace->nTracks;
    auto CurrentNvetex=wrkspace->nVertex;
    if(threadIdx.x==0)
        printf("In the updateClusterCriticalTemperatures\n");

    //      >>>>>>>>>KERNEL for T finding <<<<<<<<<
    kernel_tc_num_DF(wrkspace->tc_numer,wrkspace->zt,wrkspace->zVtx,wrkspace->pi, wrkspace->pik,wrkspace->dz2,\\
                     N,CurrentNvetex);
    __syncthreads();

    sumBlock_with_shfl_down_gid_DF(wrkspace->tc_numer,wrkspace->tc_numer,N,CurrentNvetex);
    __syncthreads();

    kernel_tc_k_DF(wrkspace->tc_numer,wrkspace->zk_denom,wrkspace->tc,N,CurrentNvetex);

}

__device__ void checkAndSplitClusters(Workspace *wrkspace)
{
    if(threadIdx.x==0)
        printf("In the checkAndSplitClusters\n");

    kernel_z_k_spliting_DF(1.0/wrkspace->beta,wrkspace->zVtx,wrkspace->rhok,wrkspace->dauterMap,wrkspace->tc,&(wrkspace->nVertex) );
    __syncthreads();

    if(threadIdx.x==0)
        printf("Number of vertices after checkAndSplitClusters = %d \n",wrkspace->nVertex);

}

__device__ void thermalize(Workspace *wrkspace,int i,size_t max_iterations_for_thermalization=20)
{
        auto N=wrkspace->nTracks;
        auto CurrentNvetex=wrkspace->nVertex;

        for(int j=0; j<max_iterations_for_thermalization; j++)
        {

            updateTrackToVertexProbablilities(wrkspace);
            __syncthreads();

            updateVertexPositions(wrkspace);
            __syncthreads();
       	   
	    updateVertexWeights(wrkspace);
            __syncthreads();

            //****** === Free Energy Monitoring   === ****************///
            if(threadIdx.x ==0 )
            {
                printf("\n\nAt i = %d , j =%d , beta =%f \n",i,j,wrkspace->beta);
                kernel_findFreeEnergyPartA<<<CurrentNvetex,N>>>(wrkspace->FEnergyA,\\
                        wrkspace->zt,wrkspace->zVtx,\\
                        wrkspace->dz2, wrkspace->beta,CurrentNvetex,N);
                kernel_findFreeEnergyPartB<<<1,1>>>(wrkspace->FEnergyA,wrkspace->beta,CurrentNvetex,N);
            }
            __syncthreads();
            //****** ===                          === ****************///

            check_ifThermalized(wrkspace->zk_delta,0.001,wrkspace->hasThermalized,CurrentNvetex);
            __syncthreads();
            if(*(wrkspace->hasThermalized)==0)
            {
                if(threadIdx.x==0)
                    printf("has thermalized for beta = %f , j =%d\n",wrkspace->beta,j);
                break;
            }
            __syncthreads();
            if(threadIdx.x==0)
                *(wrkspace->hasThermalized)=0;
        }
 
}

//__device__ void sort_vertexs(float *zVtxs,const int numberOfvertex)
//{
//    // Specialize BlockRadixSort for a 1D block of 128 threads owning 4 integer items each
//    const int NUM_ITEMS_PER_THREAD (4) ;
//    const int NUM_THREADS_FOR_RSORT(256);
//    int numThreadsInSort = numberOfvertex  / NUM_ITEMS_PER_THREAD +1 ;
//    
//    typedef hipcub::BlockRadixSort<float, NUM_THREADS_FOR_RSORT , NUM_ITEMS_PER_THREAD> BlockRadixSort;
//    // Allocate shared memory for BlockRadixSort
//    __shared__ typename BlockRadixSort::TempStorage temp_storage;
//    // Obtain a segment of consecutive items that are blocked across threads
//    
//    float tmpArray[4];
//
//    auto limit = 0;
//    if( threadIdx.x<NUM_THREADS_FOR_RSORT )
//    {
//      if(threadIdx.x<numThreadsInSort) {
//      limit=NUM_ITEMS_PER_THREAD;
//      if(threadIdx.x == numThreadsInSort-1) 
//       limit= numberOfvertex%NUM_ITEMS_PER_THREAD;
//      
//      for(auto i=0;i<limit;i++)
//        tmpArray[i]=zVtxs[threadIdx.x*NUM_ITEMS_PER_THREAD + i ];
//      
//      }
//      for(auto i=limit;i<NUM_ITEMS_PER_THREAD;i++)
//      	tmpArray[i]=1e9;
//      __syncthreads();
//      if(threadIdx.x<numThreadsInSort) {
//       
//       printf("%d thread idx -> ",threadIdx.x);
//       for(auto i=0;i<NUM_ITEMS_PER_THREAD;i++)
//          printf("%f , ",  tmpArray[i] );
//      printf("\n");
//      }
//      BlockRadixSort(temp_storage).Sort(tmpArray);
//      
//      for(auto i=0;i<limit;i++)
//          zVtxs[threadIdx.x*NUM_ITEMS_PER_THREAD + i ] = tmpArray[i];
//
//    }
//}

__device__ void checkAndMergeDaughter(float *zVtx,float *zVtx_temp,float *rhok,float *rhok_temp,int *dauterMap,float minZVtxSeparation,int currVtxCount)
{
   if(threadIdx.x==0)
        printf("In the checkIfToMergeDaughter()\n");
   
  if(threadIdx.x < currVtxCount)
    {
     auto idx=threadIdx.x;
      if(dauterMap[idx]>-1)
      {
            auto dau=dauterMap[idx];

            if(abs(zVtx[idx]-zVtx[dau])<minZVtxSeparation)
            {
                zVtx_temp[idx]      = (rhok[idx]*zVtx[idx] + rhok[dau]*zVtx[dau])/(1e-20 + rhok[dau]+rhok[idx]);
                rhok_temp[idx]      = rhok[idx]+rhok[dau];
                dauterMap[idx]      = -1;
                dauterMap[dau]      = -2;
            }
            else
            {
                zVtx_temp[idx]=zVtx[idx];
                rhok_temp[idx]=rhok[idx];
                if(dauterMap[idx]!=-2)   dauterMap[idx] = -1;
            }

#ifdef FULL_DEVICE_DEBUG
        printf("checkAndMergeDaughter DEVICE i = %d (dau = %d ): deltaZ =  %f ( <? %f )  :  mergeing zs %f,%f -> %f , rhos %f,%f -> %f \n ",\\
                    idx,dau,abs(zVtx[idx]-zVtx[dau]),minZVtxSeparation,\\
                    zVtx[idx],zVtx[dau], zVtx_temp[idx] ,\\
                    rhok[idx],rhok[dau],rhok_temp[idx]);
#endif

    }
    else
    {
    
        zVtx_temp[idx]=zVtx[idx];
        rhok_temp[idx]=rhok[idx];
        if(dauterMap[idx]!=-2)   dauterMap[idx] = -1;
    
    }
      
   }
}

__device__ void updateMergedVtxList(float *zVtx,float *rhok,float* zVtx_temp,float *rhok_temp,int *dauterMap,uint32_t currVtxCount,uint32_t *nv_tmp)
{
 
  if(threadIdx.x<currVtxCount)
    {
        
        if(dauterMap[threadIdx.x]==-2) return;
        
        auto nidx=atomicAdd(nv_tmp,1);

        zVtx[nidx]=zVtx_temp[threadIdx.x];
        rhok[nidx]=rhok_temp[threadIdx.x];
    }

}

__device__ void checkAndMergeClusters(Workspace *wrkspace)
{
    if(threadIdx.x==0)
        printf("In the checkAndMergeClusters \n");
 
    if(threadIdx.x==0)
        for(int ii=0; ii<wrkspace->nVertex; ii++)
            printf("before vertex [%d], %f \n",ii,wrkspace->zVtx[ii]);
 
    hipDeviceSynchronize();
    checkAndMergeDaughter(wrkspace->zVtx,wrkspace->zVtx_temp,wrkspace->rhok,wrkspace->rhok_temp,wrkspace->dauterMap,wrkspace->maxDZforMerge,wrkspace->nVertex);
    __syncthreads();
    hipDeviceSynchronize();
    updateMergedVtxList(wrkspace->zVtx,wrkspace->rhok,wrkspace->zVtx_temp,wrkspace->rhok_temp,wrkspace->dauterMap,wrkspace->nVertex,&(wrkspace->temp_nVertex));
 //   sort_vertexs(wrkspace->zVtx,wrkspace->nVertex);
    hipDeviceSynchronize();
    
    if(threadIdx.x==0)
    {
        wrkspace->nVertex=wrkspace->temp_nVertex;
        wrkspace->temp_nVertex=0;
    }
    if(threadIdx.x==0)
        for(int ii=0; ii<wrkspace->nVertex; ii++)
            printf("after vertex [%d], %f \n",ii,wrkspace->zVtx[ii]);
    hipDeviceSynchronize();
   
    return;
}

__global__ void dynamicSplittingPhase(Workspace * wrkspace)
{
    auto &workspace = *wrkspace;
    int i=0;

    while(workspace.beta < workspace.betaSplitMax)
    {
        // this could be avoided if we could store a sequnce of betas in the worspace precomputed
        if(threadIdx.x==0)
        {
            workspace.beta*=workspace.betaFactor;
            i+=1;
            printf("at dynamicSplittingPhase with i = %d  zt[0] = %f  , zVtx[0] = %f , beta = %f \n ",i,workspace.zt[0],workspace.zVtx[0],workspace.beta);
        }
        else i++;

        __syncthreads();
	
	thermalize(wrkspace,i,20);
        __syncthreads();

        updateClusterCriticalTemperatures(wrkspace);
        __syncthreads();

        checkAndSplitClusters(wrkspace);
        __syncthreads();
	
	thermalize(wrkspace,i,20);
        __syncthreads();

	checkAndMergeClusters(wrkspace);

        if(threadIdx.x==0)
            for(int ii=0; ii<wrkspace->nVertex; ii++)
                printf("vertex [%d] = %f \n",ii,wrkspace->zVtx[ii]);
    }

    checkAndMergeClusters(wrkspace);
    thermalize(wrkspace,i,20);
    
    if(threadIdx.x==0)
        for(int ii=0; ii<wrkspace->nVertex; ii++)
            printf("*vertex [%d], %f \n",ii,wrkspace->zVtx[ii]);
    return ;
}

__global__ void vertexAssignmentPhase(Workspace * wrkspace)
{
    auto &workspace =*wrkspace;

    // this may require some restructuring
    // there is a possibility of this to paralized with each block taking up thermalization of an individual vertex and moving the checkAndMergeClusters() to another __global__ kernel
    int i=0;
    while( i<2  /*workspace.beta < workspace.betaMax*/ )
    //while( workspace.beta < workspace.betaMax )
    {
        updateTrackToVertexProbablilities(wrkspace);
        __syncthreads();

        updateVertexPositions(wrkspace);
        __syncthreads();

        updateVertexWeights(wrkspace);
        __syncthreads();

        //checkAndMergeClusters();
        __syncthreads();

        // this could be avoided if we could store a sequnce of betas in the worspace precomputed
        if(threadIdx.x==0) {
            workspace.beta*=workspace.betaFactor;
            i++;
            printf("at vertexAssignmentPhase with i = %d , dz2[0] = %f \n",i,workspace.dz2[0]);
        }
        else i++;
        __syncthreads();

    }

    return;
}
void DAVertexer::allocateGPUworkspace()
{
    hipMalloc(&wrkspace,sizeof(Workspace));
    printf("Allocating %f  MBss on Device for Workspace\n ",sizeof(Workspace)/1e6);
}

ZVertexSoA * DAVertexer::makeAsync(ZTrackSoA * tracks,int n)
{

    //demoKernel<<<2,10>>>(tracks,n);
    //udaDeviceSynchronize();

    auto numberOfThreads = 128;
    auto numberOfBlocks  = (MAXTRACKS/numberOfThreads) + 1;

    loadTracks<<<numberOfBlocks,numberOfThreads>>>(tracks,wrkspace);
    std::cout<<(hipGetErrorName(hipGetLastError()));
    printf("\n");
    hipDeviceSynchronize();


    printf("going into initializeWorspace \n");
    initializeWorspace<<<256,1024>>>(wrkspace);
    printf("initializeWorspace returned  :  ");
    std::cout<<(hipGetErrorName(hipGetLastError()));
    printf("\n");
    hipDeviceSynchronize();

    numberOfThreads = 1024;
    initializeDAvertexReco<<<1,numberOfThreads>>>(wrkspace);
    printf("initializeDAvertexReco returned  :  ");
    std::cout<<(hipGetErrorName(hipGetLastError()));
    printf("\n");
    hipDeviceSynchronize();
    printf("numberOfThreads = %d\n",numberOfThreads);

    printf("going into dynamicSplittingPhase \n");
    numberOfThreads = 512;
    dynamicSplittingPhase<<<1,numberOfThreads>>>(wrkspace);
    printf("dynamicSplittingPhase returned  :  ");
    std::cout<<(hipGetErrorName(hipGetLastError()));
    printf("\n");
    hipDeviceSynchronize();
    printf("\n");
    printf("Out of dynamicSplittingPhase\n");

    return nullptr;

    printf("going into vertexAssignmentPhase \n");
    vertexAssignmentPhase<<<1,numberOfThreads>>>(wrkspace);
    printf("vertexAssignmentPhase returned  :  ");
    std::cout<<(hipGetErrorName(hipGetLastError()));
    printf("\n");
    std::cout<<(hipGetErrorName(hipGetLastError()));
    printf("out of vertexAssignmentPhase \n");
    hipDeviceSynchronize();
 
    printf("\n_________________________________________\n");

    return nullptr;

}

}

#include "hip/hip_runtime.h"
#include "gpuDAVertexer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"

namespace gpuDAVertexer{

__global__ void demoKernel(ZTrackSoA * tracks,int n)
{
	if(!tracks) printf("null as trks");
        int idx =threadIdx.x + blockIdx.x*blockDim.x;
	//printf("HI HI !! in device %d  idx = %d %f \n",n,idx,tracks[15].pt[0]);
	if (idx<n)
	{
	 	printf("On Devise !! [%d + %d * %d]  : track[%d].pt[0] = %f \n",threadIdx.x,blockIdx.x,blockDim.x,idx,tracks[idx].pt[0]);
	}
	else
	{
		printf("On Devise !! [%d + %d * %d] = %d\n ",threadIdx.x,blockIdx.x,blockDim.x,idx);
	}
}

__global__ void initializeWorspace(Workspace * wrkspace)
{

	auto idx= blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < MAXTRACKS*MAXVTX)
	{
		wrkspace->pik[idx]=1.0;
		wrkspace->pik_numer[idx]=0.0;
		wrkspace->pik_denom[idx]=0.0;
		
		wrkspace->zk_delta[idx]=1e8;
		wrkspace->zk_numer[idx]=0.0;
		wrkspace->zk_denom[idx]=0.0;
	}
	if(idx <MAXVTX)
	{
		wrkspace->zVtx[idx]=1e9;
	}
	if(idx==0)
	{
		wrkspace->nVertex=0;
		wrkspace->betaFactor=2.5;
	}

}

__global__ void loadTracks(ZTrackSoA * tracks,Workspace * wrkspace)
{

	auto idx= blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < tracks->ntrks)
	{
		wrkspace->zt[idx]=tracks->zt[idx];
		wrkspace->dz2[idx]=tracks->dz2[idx];
		wrkspace->pi[idx]=1.0;
	}
	if(idx==0)
	{
		if(tracks->ntrks % 2 and false)	
		{
	 	  wrkspace->zt[idx]=0.0;
		  wrkspace->dz2[idx]=1e9;
		  wrkspace->pi[idx]=0.0;
                  wrkspace->nTracks = tracks->ntrks+1; 
		}
		else
		 {
			wrkspace->nTracks=tracks->ntrks;
		 }
	}

}

	/// ==================================================//

__global__ void sumBlock_with_shfl_down_gid(float *in, float *out, int blockSize)
{ 
	int gid = blockIdx.x * blockDim.x + threadIdx.x; 
	for (int offset =  __float2uint_ru(float(blockSize)/2)  ; offset > 0; offset /= 2) 
	{        	 
	  out[gid] +=  __shfl_down_sync(0xffffffff, out[gid], offset);     
	}

 
}

__global__ void sumBlock_with_loop(float *in, float *out,int numVertices, int N)
{ 
	int tid = threadIdx.x; 
	int off ;
	out[tid]=0.0;	
	for (int offset = 0 ; offset<numVertices; offset++ )  
	{        	  
	  off = N*offset ;
	  out[tid] += in[tid+off];     
	}
//	printf("out[%d] = %f \n",tid,out[tid]);
}
__global__ void kernel_findFreeEnergyPartA(float *FEnergyA,float * zi, float *zVtx,float* sig,float beta ,int CurrentVtx,int N )
{
    int idx = threadIdx.x; 
    int bid = blockIdx.x; // block Id
    int gid = blockIdx.x * blockDim.x + idx;
    int TotalSize = N*CurrentVtx; //  nTracks * nVertex    

    if (gid < TotalSize) {
       FEnergyA[gid] =  expf( -beta*((zi[idx]-zVtx[bid])*(zi[idx]-zVtx[bid])/sig[idx] ));
//    printf("gid = %d , dz = %f - %f = %f , FEnergyA[gid] = %f\n",gid,zi[idx],zVtx[bid],zi[idx]-zVtx[bid], FEnergyA[gid]);
    }
}
__global__ void kernel_findFreeEnergyPartB(float * FEnergyA, float beta, int currVtxCount,int N)
{

	auto fEnergy=0.0;
	for(int i=0;i<N;i++)
	{
	 auto asum=0.0;
	 for(int j=0;j<currVtxCount;j++)
		{

			asum+=FEnergyA[i+j*N];
		}
//	printf("( %d , %f ,%f )",i,asum,beta);	
		fEnergy-=logf(asum >1e-20 ? asum : 1.0 )/beta;
	
	}
	printf("\n$(beta , Free Energy) : ,%f,%f\n",beta,fEnergy);
}



__global__ void kernel_p_ik_num( float *p_ik, float *z_i, float *z_k0,   float *sig, float beta, int N, int numberOfvertex )
{

    int idx = threadIdx.x; 
    int bid = blockIdx.x; // block Id
    int gid = blockIdx.x * blockDim.x + idx;
    int TotalSize = N*numberOfvertex; //  nTracks * nVertex    

    if (gid < TotalSize) {
       p_ik[gid] =  expf( -beta*(((z_i[idx]-z_k0[bid])*(z_i[idx]-z_k0[bid]))/(sig[idx]*sig[idx]*sig[idx]*sig[idx])) );
  //  printf("gid = %d , dz = %f - %f = %f , pik = %f\n",gid,z_i[idx],z_k0[bid],z_i[idx]-z_k0[bid], p_ik[gid]);
    }

}

__global__ void kernel_p_ik( float *p_ik, float *p_ik_den, int N, int numberOfvertex )
{
    
    int idx = threadIdx.x; 
    int gid = blockIdx.x * blockDim.x + idx;
    int TotalSize = N*numberOfvertex; 

    if (gid < TotalSize) 
    { 

	 auto oldval=p_ik[gid];
   	 if (p_ik_den[idx] > 1.e-45) 
   	 {   
   	     p_ik[gid] =  p_ik[gid]/p_ik_den[idx] ;
   	 }
   	 else
   	 {
   	     p_ik[gid] =  0.000 ;     
   	 }

    printf("pik[%d] = pik_[%d] / p_ik_den[%d] = %f/ %f = %f\n",\\
    		gid,gid,idx,oldval,p_ik_den[idx],p_ik[gid]);
    }

}


__global__ void kernel_z_ik_num( float *p_ik, float *z_ik_num, float *p_i, float *z_i, float *sig, int N, int numberOfvertex )
{
    int idx = threadIdx.x; 
    int gid = blockIdx.x * blockDim.x + idx;
    int TotalSize = N*numberOfvertex; //  nTracks * nVertex    
    if (gid < TotalSize) { 
        z_ik_num[gid] = p_i[idx]*p_ik[gid]*z_i[idx]/(sig[idx]*sig[idx]); 
	//printf("z_ik_num[%d] = %f ,idx  = %d , z_i[idx] = %f , p_i[idx]  = %f ,	sig[idx] = %f \n",gid,z_ik_num[gid],idx,z_i[idx],p_i[idx],sig[idx]);
    }
}


__global__ void kernel_z_ik_den( float *p_ik, float *z_ik_den, float *p_i, float *z_i, float *sig, int N, int numberOfvertex )
{
    int idx = threadIdx.x; 
    int gid = blockIdx.x * blockDim.x + idx;
    int TotalSize = N*numberOfvertex; //  nTracks * nVertex    
    if (gid < TotalSize) {  
        z_ik_den[gid] = p_i[idx]*p_ik[gid]/(sig[idx]*sig[idx]); 
	//printf("z_ik_den[%d] = %f ,idx  = %d , p_i[idx]  = %f ,	sig[idx] = %f \n",gid,z_ik_den[gid],idx,p_i[idx],sig[idx]);
    }

}

__global__ void  kernel_z_ik(float * zk_numer,float * zk_denom,float * zDelta,float* zVtx,int ntraks,int currVtxCount )
{
	if(threadIdx.x < currVtxCount)
	{
	  auto newZ=zk_numer[threadIdx.x*ntraks]/(1e-20 + zk_denom[threadIdx.x*ntraks]);
	  zDelta[threadIdx.x] = abs(zVtx[threadIdx.x] - newZ);
	  zVtx[threadIdx.x]   = newZ;
	  printf("setting Z[%d] = %f ,delta[%d] = %f ,numer = %f , deno = %f \n",\\
			threadIdx.x,zVtx[threadIdx.x],threadIdx.x,zDelta[threadIdx.x], zk_numer[threadIdx.x*ntraks],zk_denom[threadIdx.x]);
	}
}



__global__ void kernel_T0_num( float *T_num, float *z_i, float *zVtx, float *p, float *sig, int N,int currVtxCount )
{
	auto tid = threadIdx.x; 
	auto idx = threadIdx.x; 
	for(auto i=0;i<currVtxCount;i++)
	{
		idx+=N*i;
       		T_num[idx] = p[tid]*((z_i[tid]-zVtx[i])*(z_i[tid]-zVtx[i]))/(sig[tid]*sig[tid]); 
 //	printf("tid = %d, p[tid] =%f , z_i[tid] = %f ,zVtx[%d] =%f ,sig[tid] =%f , Tnum[%d] = %f \n",tid,p[tid],z_i[tid],i,zVtx[i],sig[tid],idx,T_num[idx]);
	}

	
}

__global__ void  kernel_tc_k(float * tc_numer,float * tc_denom,float* tc,int ntraks,int currVtxCount )
{
	if(threadIdx.x < currVtxCount)
	{
	  tc[threadIdx.x] = 2.0*tc_numer[threadIdx.x*ntraks]/(1e-20 + tc_denom[threadIdx.x*ntraks]);
	  printf("\n setting tc[%d] = %f , numer = %f , deno = %f \n",\\
			threadIdx.x,tc[threadIdx.x], tc_numer[threadIdx.x*ntraks],tc_denom[threadIdx.x]);
	}
}

__global__ void check_ifThermalized(float * deltas,float deltaTol ,bool &hasThermalized,int currVtxCount)
{
	hasThermalized=true;
	for(int i=0;i<currVtxCount;i++)
	{
		if(deltas[i]>deltaTol)
		{
			hasThermalized=false;
			break;
		}
	}

}

// probably pass on the z2 avg and spit approximating the xluster to be 2 gaussians
__global__ void kernel_z_k_spliting(float temp,float *z_k, float * tc_clusters ,uint32_t *cur_NV) 
{
/*  
   This kernel take the vertex list and split the last vertex into z-delta,z+delta (delta between 0 and 1.0)
*/

   auto tid= threadIdx.x;
  printf("\n\n%d , %d  \n\n",tid,*cur_NV);
   if (tid >= *cur_NV)
   	return;

   if(temp>tc_clusters[tid])
   {
    printf("Checking for vertex %d at T= %f  and Tc = %f \n ",tid,temp,tc_clusters[tid]);
	return;
   }

   auto idx =  atomicAdd(cur_NV,1);
	
   float z_k_aux =z_k [tid];

   // calculate the deltaZk 
   /*

	auto deltaZk = sqrt( <Z^2>_k - (z_k)^2 )
   */

   auto deltaZk  = abs(0.2*z_k[tid]);
   z_k[tid] = z_k_aux - deltaZk;
   z_k[idx] = z_k_aux + deltaZk;

   printf("Checking for vertex %d at T= %f  and Tc = %f, delta = %f z_old = %f z_new[%d] = %f\n ",tid,temp,tc_clusters[tid],deltaZk,z_k[tid],idx,z_k[idx]);
}



	/// =================================================//

__global__ void calculateT0(Workspace * wrkspace)
{
	if(threadIdx.x==0)
	printf("In the calculateT0 kernel \n");
}

// device functions might also be made inline , have to check if we will get any performance inprovements in this
// The calculation of Z, the Eik calculation has  space complexity of numTracks*numVertices
// In the original DA code they have arranged tracks in the acending Z and only the tracks which are close to a vertex goes into the ccalculation
// the farer tracks will only contibute very less since its supressed by exp (-Eik ) , we may have to also incorporate it after a basic working code is ready.


__global__ void initializeDAvertexReco( Workspace *wrkspace  )
{
	
	auto N=wrkspace->nTracks;
	printf("N  = %d \n",N);
	auto CurrentNvetex = 1;
	//      >>>>>>>>>KERNELs for ZVtx Update<<<<<<<<<  
	kernel_z_ik_num<<<CurrentNvetex, N>>>(wrkspace->pik, wrkspace->zk_numer, wrkspace->pi,wrkspace->zt,wrkspace->dz2, N, CurrentNvetex);
	hipDeviceSynchronize();
	kernel_z_ik_den<<<CurrentNvetex, N>>>(wrkspace->pik, wrkspace->zk_denom, wrkspace->pi, wrkspace->zt, wrkspace->dz2, N, CurrentNvetex); 
	hipDeviceSynchronize();
	sumBlock_with_shfl_down_gid<<<CurrentNvetex, N>>>(wrkspace->zk_numer, wrkspace->zk_numer, N); 
	sumBlock_with_shfl_down_gid<<<CurrentNvetex, N>>>(wrkspace->zk_denom, wrkspace->zk_denom, N);  	
	hipDeviceSynchronize();
	kernel_z_ik<<<1,CurrentNvetex>>>(wrkspace->zk_numer, wrkspace->zk_denom,wrkspace->zk_delta ,wrkspace->zVtx, N, CurrentNvetex);  

	wrkspace->nVertex=1;
 	 	
	//      >>>>>>>>>KERNEL for T finding <<<<<<<<<	
	kernel_T0_num<<<1, N>>>(wrkspace->tc_numer,wrkspace->zt,\\
					wrkspace->zVtx,wrkspace->pi ,wrkspace->dz2,\\
					N,CurrentNvetex);
	hipDeviceSynchronize();
	sumBlock_with_shfl_down_gid<<<CurrentNvetex, N>>>(wrkspace->tc_numer,wrkspace->tc_numer,N);
	// note that the denominator for Zk and Tc_k are same				
	hipDeviceSynchronize();
	kernel_tc_k<<<1,CurrentNvetex>>>(wrkspace->tc_numer,wrkspace->zk_denom,wrkspace->tc,N,CurrentNvetex);
	hipDeviceSynchronize();
	wrkspace->beta=1.0/(1e-9 + (wrkspace->tc)[0] );
	printf(" workspace beta set to %f ( 1.0/%f  , %f) \n",wrkspace->beta,wrkspace->tc[0],(wrkspace->tc)[0]);

}

__device__ void updateTrackToVertexProbablilities(Workspace * wrkspace)
{
	if(threadIdx.x==0)
	printf("In the updateTrackToVertexProbablilities\n");

//      >>>>>>>>> KERNELs for  kernel_p_ik <<<<<<<<<
	auto N=wrkspace->nTracks;
	auto CurrentNvetex=wrkspace->nVertex;
	printf("with N = %d , CurrentNvetex = %d \n",N,CurrentNvetex);
	kernel_p_ik_num<<<CurrentNvetex, N>>>(wrkspace->pik,wrkspace->zt ,wrkspace->zVtx, wrkspace->dz2, wrkspace->beta, N, CurrentNvetex);   	 
	sumBlock_with_loop <<<1,N>>> (wrkspace->pik,wrkspace->pik_denom,\\
					CurrentNvetex,N);
	kernel_p_ik<<<CurrentNvetex, N>>>(wrkspace->pik,wrkspace->pik_denom,N,CurrentNvetex);   	
	hipDeviceSynchronize();
}

__device__ void updateVertexPositions(Workspace *wrkspace)
{	
	auto N=wrkspace->nTracks;
	auto CurrentNvetex=wrkspace->nVertex;

	if(threadIdx.x==0)
	printf("In the updateVertexPositions wit %d vertexes \n",wrkspace->nVertex);
	//      >>>>>>>>>KERNELs for ZVtx Update<<<<<<<<<  
	kernel_z_ik_num<<<CurrentNvetex, N>>>(wrkspace->pik, wrkspace->zk_numer, wrkspace->pi,wrkspace->zt,wrkspace->dz2, N, CurrentNvetex);
	hipDeviceSynchronize();
	kernel_z_ik_den<<<CurrentNvetex, N>>>(wrkspace->pik, wrkspace->zk_denom, wrkspace->pi, wrkspace->zt, wrkspace->dz2, N, CurrentNvetex); 
	hipDeviceSynchronize();
	sumBlock_with_shfl_down_gid<<<CurrentNvetex, N>>>(wrkspace->zk_numer, wrkspace->zk_numer, N); 
	sumBlock_with_shfl_down_gid<<<CurrentNvetex, N>>>(wrkspace->zk_denom, wrkspace->zk_denom, N);  	
	hipDeviceSynchronize();
	kernel_z_ik<<<1,CurrentNvetex>>>(wrkspace->zk_numer, wrkspace->zk_denom,wrkspace->zk_delta ,wrkspace->zVtx, N, CurrentNvetex);  



}

__device__ void updateVertexWeights()
{
	if(threadIdx.x==0)
	printf("In the updateVertexWeights\n");
}

__device__  void updateClusterCriticalTemperatures(Workspace *wrkspace)
{
	auto N=wrkspace->nTracks;
	auto CurrentNvetex=wrkspace->nVertex;

	if(threadIdx.x==0)
	printf("In the updateClusterCriticalTemperatures\n");

	//      >>>>>>>>>KERNEL for T finding <<<<<<<<<	
	kernel_T0_num<<<1, N>>>(wrkspace->tc_numer,wrkspace->zt,\\
					wrkspace->zVtx,wrkspace->pi ,wrkspace->dz2,\\
					N,CurrentNvetex);
	hipDeviceSynchronize();
	sumBlock_with_shfl_down_gid<<<CurrentNvetex, N>>>(wrkspace->tc_numer,wrkspace->tc_numer,N);
	// note that the denominator for Zk and Tc_k are same				
	hipDeviceSynchronize();
	kernel_tc_k<<<1,CurrentNvetex>>>(wrkspace->tc_numer,wrkspace->zk_denom,wrkspace->tc,N,CurrentNvetex);
	hipDeviceSynchronize();

}

__device__ void checkAndSplitClusters(Workspace *wrkspace)
{
	if(threadIdx.x==0)
	printf("In the checkAndSplitClusters\n");
	auto CurrentNvetex = wrkspace->nVertex;

	kernel_z_k_spliting<<<1,CurrentNvetex>>>(1.0/wrkspace->beta,wrkspace->zVtx,wrkspace->tc,&(wrkspace->nVertex) );
	hipDeviceSynchronize();
	printf("Numver of vertices after checkAndSplitClusters = %d \n",wrkspace->nVertex);

}

__device__ void checkAndMergeClusters(Workspace *wrkspace)
{
	if(threadIdx.x==0)
	printf("In the checkAndMergeClusters \n");

}






__global__ void dynamicSplittingPhase(Workspace * wrkspace)
{
	auto &workspace = *wrkspace;
	int i=0;

	while(i<5 /*workspace.beta < workspace.betaSplitMax */)
	{
   // this could be avoided if we could store a sequnce of betas in the worspace precomputed
		if(threadIdx.x==0)
		  {
		  	workspace.beta*=workspace.betaFactor;
			i+=1;
			printf("at dynamicSplittingPhase with i = %d  zt[0] = %f  , zVtx[0] = %f , beta = %f \n ",i,workspace.zt[0],workspace.zVtx[0],workspace.beta);
		  }
		 else i++;

		__syncthreads();
		auto N=wrkspace->nTracks;
		auto CurrentNvetex=wrkspace->nVertex;
		
		for(int j=0;j<20;j++)
		{

			updateTrackToVertexProbablilities(wrkspace);
		        hipDeviceSynchronize(); 
			__syncthreads();
			
			updateVertexPositions(wrkspace);
		        hipDeviceSynchronize(); 
			__syncthreads();
	
			
			printf("\n\nAt i = %d , j =%d , beta =%f \n",i,j,wrkspace->beta);
			kernel_findFreeEnergyPartA<<<CurrentNvetex,N>>>(wrkspace->FEnergyA,\\
						  wrkspace->zt,wrkspace->zVtx,\\
						   wrkspace->dz2, wrkspace->beta,CurrentNvetex,N);
			kernel_findFreeEnergyPartB<<<1,1>>>(wrkspace->FEnergyA,wrkspace->beta,CurrentNvetex,N);

			check_ifThermalized<<<1,1>>>(wrkspace->zk_delta,0.001,wrkspace->hasThermalized,CurrentNvetex);
			
			hipDeviceSynchronize();

			if(wrkspace->hasThermalized)
			{
		printf("has thermalized for beta = %f , j =%d\n",wrkspace->beta,j);
				break;
			}
		}
		//updateVertexWeights(wrkspace);
		//__syncthreads();
	
	        updateClusterCriticalTemperatures(wrkspace);
		hipDeviceSynchronize();
		__syncthreads();
		checkAndSplitClusters(wrkspace);
		hipDeviceSynchronize();
		
		for(int ii=0;ii<wrkspace->nVertex;ii++)
		   printf("vertex [%d] = %f \n",ii,wrkspace->zVtx[ii]);
		}

	//checkAndMergeClusters();
	for(int ii=0;ii<wrkspace->nVertex;ii++)
		   printf("*vertex [%d], %f \n",ii,wrkspace->zVtx[ii]);
	return ;
}

__global__ void vertexAssignmentPhase(Workspace * wrkspace)
{
	auto &workspace =*wrkspace;

	// this may require some restructuring
	// there is a possibility of this to paralized with each block taking up thermalization of an individual vertex and moving the checkAndMergeClusters() to another __global__ kernel
	int i=0;
	while( i<2  /*workspace.beta < workspace.betaMax*/ )
	{
		updateTrackToVertexProbablilities(wrkspace);
		__syncthreads();
		updateVertexPositions(wrkspace);
		__syncthreads();
		updateVertexWeights();
		__syncthreads();
		//checkAndMergeClusters();
		__syncthreads();
		
   // this could be avoided if we could store a sequnce of betas in the worspace precomputed
		if(threadIdx.x==0){
		   workspace.beta*=workspace.betaFactor;
		   i++;
		   printf("at vertexAssignmentPhase with i = %d , dz2[0] = %f \n",i,workspace.dz2[0]);
		   }
		else i++;
		__syncthreads();
	
	}

	return;
}

ZVertexSoA * DAVertexer::makeAsync(ZTrackSoA * tracks,int n)
{
 
	 Workspace *wrkspace;
	 hipMalloc(&wrkspace,sizeof(Workspace));
	
	 //demoKernel<<<2,10>>>(tracks,n);
	 //udaDeviceSynchronize(); 
	 
	 auto numberOfThreads = 128;
	 auto numberOfBlocks  = (MAXTRACKS/numberOfThreads) + 1;
	
	 loadTracks<<<numberOfBlocks,numberOfThreads>>>(tracks,wrkspace);
	 hipDeviceSynchronize(); 
	 
	 
	 initializeWorspace<<<256,1024>>>(wrkspace);
	 initializeDAvertexReco<<<1,1>>>(wrkspace);
         hipDeviceSynchronize(); 
	 printf("\n");
	

	 dynamicSplittingPhase<<<1,1>>>(wrkspace);
	 hipDeviceSynchronize(); 
	 printf("\n");
	 
	 return nullptr;
	 
	 vertexAssignmentPhase<<<1,102>>>(wrkspace);
	 printf("\n");
	 
	 //printf(hipGetErrorName(hipGetLastError()));
	 hipDeviceSynchronize(); 
	 printf("\n");
	
	 return nullptr;

}

}

#include "hip/hip_runtime.h"
#include "gpuDAVertexer.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "stdio.h"

__global__ void demoKernel(ZTrackSoA * tracks,int n)
{
	if(!tracks) printf("null as trks");
        int idx =threadIdx.x + blockIdx.x*blockDim.x;
	//printf("HI HI !! in device %d  idx = %d %f \n",n,idx,tracks[15].pt[0]);
	if (idx<n)
	{
	 	printf("On Devise !! [%d + %d * %d]  : track[%d].pt[0] = %f \n",threadIdx.x,blockIdx.x,blockDim.x,idx,tracks[idx].pt[0]);
	}
	else
	{
		printf("On Devise !! [%d + %d * %d] = %d\n ",threadIdx.x,blockIdx.x,blockDim.x,idx);
	}
}

ZVertexSoA * gpuDAVertexer::DAVertexer::makeAsync(ZTrackSoA * tracks,int n)
{
  printf("\n in the makeAsync n = %d \n",n);	  
  demoKernel<<<2,10>>>(tracks,n);
  hipDeviceSynchronize(); 
  return nullptr;
}

